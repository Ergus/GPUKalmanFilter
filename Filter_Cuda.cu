#include "hip/hip_runtime.h"
#include "Filter_Cuda.h"

__device__ float filter(const float z, float *x, float *tx,
                    float *covXX,float *covXTx,
                    float *covTxTx, const float zhit,
                    const float xhit, const float whit) {
    // compute the prediction
    const float dz = zhit - z;
    const float predx = (*x) + dz * (*tx);

    const float dz_t_covTxTx = dz * (*covTxTx);
    const float predcovXTx = (*covXTx) + dz_t_covTxTx;
    const float dx_t_covXTx = dz * (*covXTx);

    const float predcovXX =(*covXX)+2*dx_t_covXTx+dz*dz_t_covTxTx;
    const float predcovTxTx = (*covTxTx);
    // compute the gain matrix
    const float R = 1.0 / (1.0 / whit + predcovXX);
    const float Kx = predcovXX * R;
    const float KTx = predcovXTx * R;
    // update the state vector
    const float r = xhit - predx;
    *x = predx + Kx * r;
    *tx = (*tx) + KTx * r;
    // update the covariance matrix.
    *covXX = (1 - Kx) * predcovXX;
    *covXTx = (1 - Kx) * predcovXTx;
    *covTxTx = predcovTxTx - KTx * predcovXTx;
    // return the chi2
    return r*r*R;
    }

#if UCUDA == 1
__global__ void Kalman_Filter(float* ttrack,
                            int* trstart,
                            float* fullin,
                            int* backward,
                            float* sum2,
                            float* fullout,
                            const unsigned int tracks,
                            const unsigned int hits){

    //Declared before because it is the most important var
    const int idx = blockIdx.x*blockDim.x+threadIdx.x;

    if(idx>=tracks) return;
    
    #if (defined DEBUG && __CUDA_ARCH__ >= 200)
    if(idx==0)
        printf("Using kernel 1 for Cuda for %d tracks\n",tracks);
    #endif
    
    int first = trstart[idx],
        last = trstart[idx+1],
        direction = (backward[idx] ? 1 : -1),
        dhit, size=last-first-1;
    const float noise2PerLayer=sum2[idx];
    
    float ax[24],ay[24],
        az[24],aerrx[24],aerry[24];

    for(int i=first, j=0;i<last;i++, j++){
        ax[j]   = fullin[i];
        ay[j]   = fullin[i+hits];
        az[j]   = fullin[i+2*hits];
        aerrx[j]= fullin[i+3*hits];
        aerry[j]= fullin[i+4*hits];        
        }

    if((az[size]-az[0])*direction<0){
        dhit=-1;
        first=size;
        last=-1;
        }
    else{
        dhit=1;
        last=size+1;
        first=0;
        }
    
    float x = ax[first],
        tx = ttrack[idx],
         y = ay[first],
        ty = ttrack[idx+tracks],
         z = az[first],
        wx = aerrx[first],
        wy = aerry[first];
    
    // initialize the covariance matrix
    float covXTx  = 0.0f;  // no initial correlation
    float covYTy  = 0.0f;
    float covTxTx = 1.0f;  // randomly large error
    float covTyTy = 1.0f;
    float covXX = 1.0f /(wx*wx);
    float covYY = 1.0f /(wy*wy);
    float chi2=0.0f;
    
    for (int i=first+dhit; i!=last; i+=dhit) {    
        wx=aerrx[i];
        wy=aerry[i];
        covTxTx+=noise2PerLayer;
        covTyTy+=noise2PerLayer;

        // filter X
        chi2 += filter(z, &x, &tx, &covXX, &covXTx, &covTxTx,az[i], ax[i], wx*wx);
        // filter Y
        chi2 += filter(z, &y, &ty, &covYY, &covYTy, &covTyTy,az[i], ay[i], wy*wy);
        z=az[i];
        }

    // add the noise at the last hit
    covTxTx += noise2PerLayer;
    covTyTy += noise2PerLayer;

    // finally, fill the state
    int tmp=11*idx;
    fullout[tmp+0]=x;
    fullout[tmp+1]=y;
    fullout[tmp+2]=z;
    fullout[tmp+3]=tx;
    fullout[tmp+4]=ty;

    fullout[tmp+5] = covXX;
    fullout[tmp+6] = covXTx;
    fullout[tmp+7] = covTxTx;
    fullout[tmp+8] = covYY;
    fullout[tmp+9] = covYTy;
    fullout[tmp+10]= covTyTy;
    
    }

#elif UCUDA == 2
__global__ void Kalman_Filter(float* ttrack,
                              int* trstart,
                              float* fullin,
                              int* backward,
                              float* sum2,
                              float* fullout,
                              const unsigned int tracks,
                              const unsigned int hits){

    //Declared before because it is the most important var
    const int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx>=tracks) return;
    
    const int idy = blockIdx.y*blockDim.y+threadIdx.y;
    #if (defined DEBUG && __CUDA_ARCH__ >= 200)
    if((idx==0) && (idy==0))
        printf("Using kernel 2 for Cuda for %d tracks\n",tracks);
    #endif
    
    int first = trstart[idx],
        last = trstart[idx+1],
        direction = (backward[idx] ? 1 : -1),
        dhit, size=last-first-1;
    const float noise2PerLayer=sum2[idx];
    
    float ax[24],az[24],aerrx[24];

    for(int i=first, j=0;i<last;i++, j++){
        ax[j]   = fullin[i+idy*hits];
        az[j]   = fullin[i+2*hits];
        aerrx[j]= fullin[i+(idy+3)*hits];
        }

    if((az[size]-az[0])*direction<0){
        dhit=-1;
        first=size;
        last=-1;
        }
    else{
        dhit=1;
        last=size+1;
        first=0;
        }
    
    float x = ax[first],
        tx = ttrack[idx],
         z = az[first],
        wx = aerrx[first];
    
    // initialize the covariance matrix
    float covXTx  = 0.0f;  // no initial correlation
    float covTxTx = 1.0f;  // randomly large error
    float covXX = 1.0f /(wx*wx);
    float chi2=0.0f;
    
    for (int i=first+dhit; i!=last; i+=dhit) {    
        wx=aerrx[i];
        covTxTx+=noise2PerLayer;

        // filter X
        chi2 += filter(z, &x, &tx, &covXX, &covXTx, &covTxTx,az[i], ax[i], wx*wx);
        // filter Y
        z=az[i];
        }

    // add the noise at the last hit
    covTxTx += noise2PerLayer;

    // finally, fill the state
    int tmp=11*idx;
    fullout[tmp+idy]=x;
    if(idy==0) fullout[tmp+2]=z;
    fullout[tmp+3+idy]=tx;

    fullout[tmp+5+3*idy] = covXX;
    fullout[tmp+6+3*idy] = covXTx;
    fullout[tmp+7+3*idy] = covTxTx;
    
    }
#endif


float cudaFilter(int *evstart,
               int *trstart,
               float *ttrack,
               float *fullin,
               int *backward,
               float *sum2,
               float *fullout,
               size_t events,
               size_t tracks,
               size_t hits){


    hipEvent_t gpu_start, gpu_stop;
    double cpu_start, cpu_stop;
    float gpu_time;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    
    dim3 dimBlock(LOCALSIZE,UCUDA),
         dimGrid( (tracks+LOCALSIZE-1)/LOCALSIZE,1);

    int *dev_evstart, *dev_trstart, *dev_backward;
    float *dev_ttrack, *dev_fullin, *dev_sum2, *dev_fullout;

    //=======Allocate memory in the device==================
    hipMalloc((void**)&dev_evstart, (events+1)*sizeof(int)); //Array int start events [nbevents+1]
    cudaCheck(hipMalloc((void**)&dev_trstart, (tracks+1)*sizeof(int))); //Array int start tracks [nbtracks+1]
    cudaCheck(hipMalloc((void**)&dev_backward, (tracks)*sizeof(int)));  //Array bool backward    [nbtracks]

    cudaCheck(hipMalloc((void**)&dev_ttrack, 2*tracks*sizeof(float)));  //Array for tx,ty/track  [2*tracks]
    cudaCheck(hipMalloc((void**)&dev_fullin, 5*hits*sizeof(float)));    //Array float data hits  [5*nbhits]
    cudaCheck(hipMalloc((void**)&dev_sum2, tracks*sizeof(float)));      //Array float parameter  [nbtracks]
    cudaCheck(hipMalloc((void**)&dev_fullout, 11*tracks*sizeof(float)));//Array float results.   [11*nbtracks]

    //=======Copy arrays to the device======================
    cudaCheck(hipMemcpy( dev_evstart, evstart, (events+1)*sizeof(int), hipMemcpyHostToDevice ));
    cudaCheck(hipMemcpy( dev_trstart, trstart, (tracks+1)*sizeof(int), hipMemcpyHostToDevice ));
    cudaCheck(hipMemcpy( dev_backward, backward, tracks*sizeof(int), hipMemcpyHostToDevice ));
    
    cudaCheck(hipMemcpy( dev_ttrack, ttrack, 2*tracks*sizeof(float), hipMemcpyHostToDevice ));
    cudaCheck(hipMemcpy( dev_fullin, fullin, 5*hits*sizeof(float), hipMemcpyHostToDevice ));
    cudaCheck(hipMemcpy( dev_sum2, sum2, tracks*sizeof(float), hipMemcpyHostToDevice ));
    
    //----------------------------
    hipEventRecord(gpu_start, 0);
    cpu_start=mtimes();
    Kalman_Filter<<<dimGrid,dimBlock>>>(dev_ttrack,
                                     dev_trstart,
                                     dev_fullin,
                                     dev_backward,
                                     dev_sum2,
                                     dev_fullout,
                                     tracks,
                                     hits);
    hipDeviceSynchronize();
    cpu_stop=mtimes();
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_time, gpu_start, gpu_stop);
        
    hipMemcpy(fullout, dev_fullout, 11*tracks*sizeof(float), hipMemcpyDeviceToHost);

    printf("Kernel execution time GPU= %0.3f ns\n", gpu_time*1000);
    printf("Kernel execution time CPU= %0.3lf ns\n", (cpu_stop-cpu_start)*1.0E6);

    hipFree(dev_ttrack);
    hipFree(dev_evstart);    
    hipFree(dev_trstart);
    hipFree(dev_fullin);
    hipFree(dev_backward);
    hipFree(dev_sum2);
    hipFree(dev_fullout);
    return 0;
    }
